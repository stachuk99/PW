
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

__global__ void matrixMultiply(int *matrix1, int *matrix2, int *matrix3, int m, int p) {
    int sum=0;
    int i =  blockIdx.x*64 + threadIdx.x; 
    for (int j = 0; j < p; j++) 
    {
        for (int k = 0; k < m; k++) 
        {
            sum = sum + matrix1[m*i+k]*matrix2[p*k+j];
        }
        matrix3[p*i+j] = sum;
        sum = 0;
     }    
}

int main(int argc, char **argv)
{   
    clock_t start_cpu,stop_cpu,start_gpu,stop_gpu;
    srand(time(NULL));
    int n = 0,m,p;
    if (argc>1) {n=strtol(argv[1],NULL,10);}
    m=n;
    p=n;
    //if (argc>2) {m=strtol(argv[2],NULL,10);}
    //if (argc>3) {p=strtol(argv[3],NULL,10);}
    //printf("n: %d, m:%d, p:%d\n",n,m,p);
    int *mat1 = (int *)malloc(n*m*sizeof(int));
    int *mat2 = (int *)malloc(m*p*sizeof(int));
    int *mat3 = (int *)malloc(n*p*sizeof(int));
        
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < m; j++)
        {
            mat1[m*i+j] = (int) rand()%5;
        }
    }
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < p; j++)
        {
            mat2[p*i+j] = (int) rand()%5;
        }
    }
    int *c_mat1;
    hipMalloc(&c_mat1,n*m*sizeof(int));
    int *c_mat2;
    hipMalloc(&c_mat2,m*p*sizeof(int));
    int *c_mat3;
    hipMalloc(&c_mat3,n*p*sizeof(int));
    hipMemcpy(c_mat1, mat1, n*m*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(c_mat2, mat2, m*p*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(c_mat3, mat3, n*p*sizeof(int), hipMemcpyHostToDevice);
    start_cpu = clock();
    int sum=0;
    for (int i = 0; i < n; i++) 
    {
        for (int j = 0; j < p; j++) 
        {
            for (int k = 0; k < m; k++) 
            {
                sum = sum + mat1[m*i+k]*mat2[p*k+j];
            }
            mat3[p*i+j] = sum;
            //printf("%d, ",mat3[p*i+j]);
            sum = 0;
        }
        //printf("\n");
    }
    stop_cpu = clock();
    int time_spent = (int)(1000 * (stop_cpu - start_cpu) / CLOCKS_PER_SEC);
    printf("%d  ",time_spent);
    start_gpu = clock();
    int bloki = n / 64;
    int watki =  64;
    if (n%64 != 0 ) bloki++;   
    matrixMultiply<<<bloki,watki>>>(c_mat1,c_mat2,c_mat3,m,p);
    hipDeviceSynchronize();
    stop_gpu = clock();
    int time_spent2 = (int)(1000 * (stop_gpu - start_gpu) / CLOCKS_PER_SEC);
    printf("%d\n",time_spent2);
    hipMemcpy(mat3, c_mat3, n*p*sizeof(int), hipMemcpyDeviceToHost);
    /*for (int i = 0; i < n; i++) 
    {
        for (int j = 0; j < p; j++) 
        {
            printf("%d, ",mat3[p*i+j]);
        }
        printf("\n");
    }*/
 return 0;
}

